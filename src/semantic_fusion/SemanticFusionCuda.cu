/*
 * This file is part of SemanticFusion.
 *
 * Copyright (C) 2017 Imperial College London
 * 
 * The use of the code within this file and all code within files that 
 * make up the software that is SemanticFusion is permitted for 
 * non-commercial purposes only.  The full terms and conditions that 
 * apply to the code within this file are detailed within the LICENSE.txt 
 * file and at <http://www.imperial.ac.uk/dyson-robotics-lab/downloads/semantic-fusion/semantic-fusion-license/> 
 * unless explicitly stated.  By downloading this file you agree to 
 * comply with these terms.
 *
 * If you wish to use any of this code for commercial purposes then 
 * please email researchcontracts.engineering@imperial.ac.uk.
 *
 */

#include <stdio.h>
#include <assert.h> 

#include <hip/hip_runtime.h>

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool
        abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    } 
}

__global__ 
void semanticTableUpdate(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probabilities, const int prob_width, const int prob_height, 
                          const int prob_channels,float* map_table,float* map_max,
                          const int map_size)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    // New uniqueness code
    const int check_patch = 16;
    const int x_min = (x - check_patch) < 0 ? 0 : (x - check_patch);
    const int x_max = (x + check_patch) > 640 ? 640 : (x + check_patch);
    const int y_min = (y - check_patch) < 0 ? 0 : (y - check_patch);
    const int y_max = (y + check_patch) < 480 ? 480 : (y + check_patch);

    int surfel_id = tex2D<int>(ids,x,y);
    int first_h, first_w;
    // for (int h = y_min; h < 480; ++h) {
    //     for (int w = x_min; w < x_max; ++w) {
    //         int other_surfel_id = tex2D<int>(ids,w,h);
    //         if (other_surfel_id == surfel_id) {
    //             first_h = h;
    //             first_w = w;
    //             break;
    //         }
    //     }
    // }

    //patch rule 1
    // for (int h = y_min; h < 480; ++h) {
    //     int other_surfel_id;
    //     for (int w = x_min; w < x_max; ++w) {
    //         other_surfel_id = tex2D<int>(ids,w,h);
    //         if (other_surfel_id == surfel_id) {
    //             first_h = h;
    //             first_w = w;
    //             break;
    //         }
    //     }
    //     if (other_surfel_id == surfel_id) {
    //             break;
    //         }
    // }
    //patch rule 2
    for (int h = y_min; h < y_max; ++h) {
        int other_surfel_id;
        for (int w = x_min; w < x_max; ++w) {
            other_surfel_id = tex2D<int>(ids,w,h);
            if (other_surfel_id == surfel_id) {
                first_h = h;
                first_w = w;
                break;
            }
        }
        if (other_surfel_id == surfel_id) {
                break;
            }
    }

    if (first_h != y || first_w != x) {
        surfel_id = 0;
    }
    if (surfel_id > 0) {
        // x,y coordinates in probability image
        const int prob_x = static_cast<int>((float(x) / ids_width) * prob_width);
        const int prob_y = static_cast<int>((float(y) / ids_height) * prob_height);
        
        // memory offset of the probability of the neighborhood class at the same pixel of probability image
        const int channel_offset = prob_width * prob_height; 
        
        // pointer at (prob_x,prob_y)
        const float* probability = probabilities + (prob_y * prob_width + prob_x);

        // pointer at the surfel in prob_table
        float* prior_probability = map_table + surfel_id;

        // go though all class channels to update prob of the correspond surfel
        float total = 0.0;
        for (int class_id = 0; class_id < prob_channels; ++class_id) {
            prior_probability[0] *= probability[0]; // use prob of a class of a pixel to update its correponsded surfel
            total += prior_probability[0];  // sum prob of all classes
            probability += channel_offset;  // go to the next class prob on prob image
            prior_probability += map_size;  // go to the next class prob on surfel map
        }

        // Reset the pointers to the beginning again
        probability = probabilities + (prob_y * prob_width + prob_x);
        prior_probability = map_table + surfel_id;
        float max_probability = 0.0;
        int max_class = -1;
        float new_total = 0.0;
        // normalize probs and search the class with max prob
        for (int class_id = 0; class_id < prob_channels; ++class_id) {
            // Something has gone unexpectedly wrong - reinitialse
            if (total <= 1e-5) {
                prior_probability[0] = 1.0f / prob_channels;
            } else {
                prior_probability[0] /= total; // normalize prob 
                if (class_id > 0 && prior_probability[0] > max_probability) {
                    max_probability = prior_probability[0];
                    max_class = class_id;
                }
            }
            new_total += prior_probability[0];
            probability += channel_offset;
            prior_probability += map_size;
        }
        map_max[surfel_id] = static_cast<float>(max_class);
        map_max[surfel_id + map_size] = max_probability;
        map_max[surfel_id + map_size + map_size] += 1.0;
    }
}

__host__ 
void fuseSemanticProbabilities(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probabilities, const int prob_width, const int prob_height, 
                          const int prob_channels,float* map_table, float* map_max,
                          const int map_size)
{
    // NOTE Res must be pow 2 and > 32
    const int blocks = 32;
    dim3 dimGrid(blocks,blocks);
    dim3 dimBlock(640/blocks,480/blocks);
    semanticTableUpdate<<<dimGrid,dimBlock>>>(ids,ids_width,ids_height,probabilities,prob_width,prob_height,prob_channels,map_table,map_max,map_size);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}

__global__ 
void updateTable(int n, const int* deleted_ids, const int num_deleted, const int current_table_size,
                 float const* probability_table, const int prob_width, const int prob_height, 
                 const int new_prob_width, float* new_probability_table, float const * map_table, float* new_map_table)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;  // kernal index
    if (index < n) {
        const int class_id = index / new_prob_width;  // get class id of current kernal in new table
        const int component_id = index - (class_id * new_prob_width);  // get surfel id of current kernal in new table
        const int new_id = (class_id * prob_width) + component_id; // get table index with max_componets as width
        if (component_id >= num_deleted) {
            // Initialise to prior (prob height is the number of classes)
            new_probability_table[new_id] = 1.0f / prob_height;
            // Reset the max class surfel colouring lookup
            new_map_table[component_id] = -1.0;
            new_map_table[component_id + prob_width] = -1.0;
            new_map_table[component_id + prob_width + prob_width] = 0.0;
        } else {
            int offset = deleted_ids[component_id]; // get corresponded surf_id in previous table
            new_probability_table[new_id] = probability_table[(class_id * prob_width) + offset];
            // Also must update our max class mapping
            new_map_table[component_id] = map_table[offset];
            new_map_table[component_id + prob_width] = map_table[prob_width + offset];
            new_map_table[component_id + prob_width + prob_width] = map_table[prob_width + prob_width + offset];
        }
    }
}

__host__ 
void updateProbabilityTable(int* filtered_ids, const int num_filtered, const int current_table_size,
                            float const* probability_table, const int prob_width, const int prob_height, 
                            const int new_prob_width, float* new_probability_table, 
                            float const* map_table, float* new_map_table)
/*
filtered_ids: map->GetDeletedSurfelIdsGpu(),
num_filtered: num_deleted,
current_table_size: current_table_size_,
probability_table: class_probabilities_gpu_->gpu_data(),
prob_width: table_width, prob_height: table_height,
new_prob_width: new_table_width, 
new_probability_table: class_probabilities_gpu_buffer_->mutable_gpu_data(),
map_table: class_max_gpu_->gpu_data(),
new_map_table: class_max_gpu_buffer_->mutable_gpu_data()
*/
{   

    const int threads = 512;
    const int num_to_update = new_prob_width * prob_height; // new_table_width*num_classes_
    const int blocks = (num_to_update + threads - 1) / threads;  
    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    updateTable<<<dimGrid,dimBlock>>>(num_to_update,filtered_ids,num_filtered,current_table_size,probability_table,prob_width,prob_height,new_prob_width,new_probability_table, map_table, new_map_table);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}


__global__ 
void renderProbabilityMapKernel(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probability_table, const int prob_width, const int prob_height, 
                          float* rendered_probabilities) 
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int surfel_id = tex2D<int>(ids,x,y);
    int projected_probability_offset = y * ids_width + x;
    int probability_table_offset = surfel_id;
    for (int class_id = 0; class_id < prob_height; ++class_id) {
        if (surfel_id > 0) {
            rendered_probabilities[projected_probability_offset] = probability_table[probability_table_offset];
        } else {
            rendered_probabilities[projected_probability_offset] = ((class_id == 0) ? 1.0 : 0.0);
        }
        projected_probability_offset += (ids_width * ids_height);
        probability_table_offset += prob_width;
    }
}


__host__
void renderProbabilityMap(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probability_table, const int prob_width, const int prob_height, 
                          float* rendered_probabilities) 
{
    // NOTE Res must be pow 2 and > 32
    const int blocks = 32;
    dim3 dimGrid(blocks,blocks);
    dim3 dimBlock(ids_width/blocks,ids_height/blocks);
    renderProbabilityMapKernel<<<dimGrid,dimBlock>>>(ids,ids_width,ids_height,probability_table,prob_width,prob_height,rendered_probabilities);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}

__global__ 
void updateMaxClassKernel(const int n, const float* probabilities, const int classes,
                          float* map_max, const int map_size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        // Reset the pointers to the beginning again
        const float* probability = probabilities + index;
        probability += map_size;
        float max_probability = 0.0;
        int max_class = -1;
        for (int class_id = 1; class_id < classes; ++class_id) {
            if (probability[0] > max_probability) {
                max_probability = probability[0];
                max_class = class_id;
            }
            probability += map_size;
        }
        map_max[index] = static_cast<float>(max_class);
        map_max[index + map_size] = max_probability;
    }
}

__host__ 
void updateMaxClass(const int n, const float* probabilities, const int classes,
                    float* map_max, const int map_size)
{
    const int threads = 512;
    const int blocks = (n + threads - 1) / threads;
    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    updateMaxClassKernel<<<dimGrid,dimBlock>>>(n,probabilities,classes,map_max,map_size);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}
